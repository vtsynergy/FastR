#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include "FastOffset.h"


#define GetreadSeq(a) &(readSeq[a * 100])
#define Getquals(a) &(quals[a * 100])
#define byte char
#define MAX_QUAL 99
#define MAX_ReadLength 100

#define byte char
byte *dreadSeq;
byte *dquals;
byte *dref;
int *dScoreIndex;
int *dBestScore;
int *dReadLength;
int *doriginalAlignment;
int *dn_consensus;
int *daltReads;
int *drefLens;
char *dch_lookup;
extern int score_offset;

using namespace std;
using namespace BamTools;

hipError_t cudaRes;

#define CHECKERR(cudaRes)\
{\
    if (cudaRes != hipSuccess)\
    printf("cuda allocate on device error: %d in line:%d (%s)\n", cudaRes, __LINE__, __FILE__);\
}

#define CHECKERR1()\
{\
    cudaRes = hipGetLastError();\
    if(cudaRes != hipSuccess)\
    printf("hipError_t: %d %s in line:%d (%s) \n", cudaRes, hipGetErrorString(cudaRes), __LINE__, __FILE__);\
}


__device__ int mismatchQualitySumIgnoreCigar2_CUDA(byte *readSeq, byte *quals, byte *refSeq, int refLen, int refIndex, int getReadLength, char *ch_lookup, int quitAboveThisValue)
{
    int sum = 0;
    for(int readIndex = 0 ; readIndex < getReadLength; refIndex++, readIndex++) {
        if(refIndex >= refLen) {
            sum += MAX_QUAL;
            if(sum > quitAboveThisValue) {
                return sum;
            }
        } else {
            byte refChr = refSeq[refIndex];
            byte readChr = readSeq[readIndex];
            if(!ch_lookup[readChr] || !ch_lookup[refChr]) {
                continue;    // do not count Ns/Xs/etc ?
            }
            if((readChr != refChr)) {
                sum += (int)quals[readIndex];
                if(sum > quitAboveThisValue) {
                    return sum;
                }
            }
        }
    }
    return sum;
}


__device__ int mismatchQualitySumIgnoreCigar_CUDA(byte *readSeq, byte *quals, byte *refSeq, int refLen, int refIndex, int getReadLength, char *ch_lookup)
{
    int sum = 0;
    for(int readIndex = 0 ; readIndex < getReadLength; refIndex++, readIndex++) {
        if(refIndex >= refLen) {
            sum += MAX_QUAL;
        } else {
            byte refChr = refSeq[refIndex];
            byte readChr = readSeq[readIndex];
            if(!ch_lookup[readChr] || !ch_lookup[refChr]) {
                continue;    // do not count Ns/Xs/etc ?
            }
            if((readChr != refChr)) {
                sum += (int)quals[readIndex];
            }
        }
    }
    return sum;
}

__global__ void findBestOffset_CUDA(byte *ref_g,  byte *readSeq_g,  byte *quals_g,  int *ReadLength_g,  int *originalAlignment_g,  int *bestScore_g,
                                    int *bestIndex_g,  char *ch_lookup, int iter_num,  int *n_consensus, int *altReads,  int *refLens)
{
    int b = blockIdx.x;
    int i = threadIdx.x;
    int ii = threadIdx.x + blockDim.x;
    __shared__ int score[1024];
    __shared__ int scoreindex[1024];
    int ref_offset = 0;
    int quals_offset = 0;
    int ReadLength_offset = 0;
    int originalAlignment_offset = 0;
    int score_offset = 0;
    int n_refLen = 0;
    int readSeq_offset = 0;

    for(int t = 0; t < iter_num; t++) {
        for(int k = 0; k < n_consensus[t]; k++) {
            if(b < altReads[t]) {

                score[i] = INT_MAX;
                score[ii] = INT_MAX;
                scoreindex[i] = i;
                scoreindex[ii] = ii;

                byte *ref = ref_g + ref_offset;
                byte *readSeq = readSeq_g + readSeq_offset;
                byte *quals = quals_g + quals_offset;
                int *ReadLength = ReadLength_g + ReadLength_offset;
                int *originalAlignment = originalAlignment_g + originalAlignment_offset;
                int *bestScore = bestScore_g + score_offset;
                int *bestIndex = bestIndex_g + score_offset;

                if(i == 0) {
                    bestScore[b] = mismatchQualitySumIgnoreCigar_CUDA(GetreadSeq(b), Getquals(b), ref, refLens[n_refLen], originalAlignment[b], ReadLength[b], ch_lookup);
                    bestIndex[b] = originalAlignment[b];
                }

                __syncthreads();

                if(i < originalAlignment[b]) {
                    score[i] = mismatchQualitySumIgnoreCigar2_CUDA(GetreadSeq(b), Getquals(b), ref, refLens[n_refLen], i, ReadLength[b], ch_lookup, bestScore[b]);
                }
                if(ii < originalAlignment[b]) {
                    score[ii] = mismatchQualitySumIgnoreCigar2_CUDA(GetreadSeq(b), Getquals(b), ref, refLens[n_refLen], ii, ReadLength[b], ch_lookup, bestScore[b]);
                }

                __syncthreads();

                for(unsigned int s = blockDim.x / 2; s > 0; s >>= 1) {
                    if(i < s) {
                        if(score[i] > score[i + s] || (scoreindex[i + s] < scoreindex[i] && score[i] == score[i + s])) {
                            score[i] = score[i + s];
                            scoreindex[i] = scoreindex[i + s];
                        }
                        if(score[ii] > score[ii + s] || (scoreindex[ii + s] < scoreindex[ii] && score[ii] == score[ii + s])) {
                            score[ii] = score[ii + s];
                            scoreindex[ii] = scoreindex[ii + s];
                        }
                    }
                    __syncthreads();
                }

                if(i == 0) {
                    if(score[0] > score[blockDim.x] || (scoreindex[blockDim.x] < scoreindex[0] && score[0] ==  score[blockDim.x])) {
                        score[0] = score[blockDim.x];
                        scoreindex[0] = scoreindex[blockDim.x];
                    }
                    if(bestScore[b] > score[0]) {
                        bestScore[b] = score[0];
                        bestIndex[b] = scoreindex[0];
                    }
                }

                __syncthreads();

                score[i] = INT_MAX;
                scoreindex[i] = i ;
                score[ii] = INT_MAX;
                scoreindex[ii] = ii ;
                int maxPossibleStart = refLens[n_refLen] - ReadLength[b];
                maxPossibleStart = maxPossibleStart - (originalAlignment[b] + 1);

                if(i <= maxPossibleStart) {
                    score[i] = mismatchQualitySumIgnoreCigar2_CUDA(GetreadSeq(b), Getquals(b), ref, refLens[n_refLen], i + originalAlignment[b] + 1, ReadLength[b], ch_lookup, bestScore[b]);
                }
                if(ii <= maxPossibleStart) {
                    score[ii] = mismatchQualitySumIgnoreCigar2_CUDA(GetreadSeq(b), Getquals(b), ref, refLens[n_refLen], ii + originalAlignment[b] + 1, ReadLength[b], ch_lookup, bestScore[b]);
                }

                __syncthreads();

                for(unsigned int s = blockDim.x / 2; s > 0; s >>= 1) {
                    if(i < s) {
                        if(score[i] > score[i + s] || (scoreindex[i + s] < scoreindex[i] && score[i] == score[i + s])) {
                            score[i] = score[i + s];
                            scoreindex[i] = scoreindex[i + s];
                        }
                        if(score[ii] > score[ii + s] || (scoreindex[ii + s] < scoreindex[ii] && score[ii] == score[ii + s])) {
                            score[ii] = score[ii + s];
                            scoreindex[ii] = scoreindex[ii + s];
                        }
                    }
                    __syncthreads();
                }

                if(i == 0) {
                    if(score[0] > score[blockDim.x] || (scoreindex[blockDim.x] < scoreindex[0] && score[0] ==  score[blockDim.x])) {
                        score[0] = score[blockDim.x];
                        scoreindex[0] = scoreindex[blockDim.x];
                    }
                    if(bestScore[b] > score[0]) {
                        bestScore[b] = score[0];
                        bestIndex[b] = scoreindex[0] + originalAlignment[b] + 1;
                    }
                }
            }
            ref_offset += refLens[n_refLen];
            n_refLen += 1;
            score_offset += altReads[t];
        }
        readSeq_offset += altReads[t] * MAX_ReadLength;
        quals_offset += MAX_ReadLength * altReads[t];
        ReadLength_offset += altReads[t];
        originalAlignment_offset += altReads[t];
    }
}

void cpy2d_lookup(char *ch_lookup)
{
    hipError_t cudaRes0;
    cudaRes0 = hipMalloc((void **)&dch_lookup, sizeof(char) * 128);
    CHECKERR(cudaRes0);
    hipMemcpy(dch_lookup, ch_lookup, sizeof(char) * 128, hipMemcpyHostToDevice);
}

void free_lookup()
{
    hipError_t cudaRes;
    hipFree(dch_lookup);
    CHECKERR1();
}

void free_resource()
{
    hipError_t cudaRes;
    hipFree(dn_consensus);
    hipFree(daltReads);
    hipFree(drefLens);
    hipFree(dScoreIndex);
    hipFree(dBestScore);
    hipFree(dref);
    hipFree(dreadSeq);
    hipFree(dquals);
    hipFree(dReadLength);
    hipFree(doriginalAlignment);
    CHECKERR1();
}



void clean_cuda(int iter_num, const size_t globalWorkSize,  const size_t localWorkSize, int *BestScore, int *ScoreIndex)
{
    hipError_t cudaRes;
    findBestOffset_CUDA <<< globalWorkSize, localWorkSize>>> (dref,  dreadSeq,  dquals, dReadLength,  doriginalAlignment,
            dBestScore, dScoreIndex,  dch_lookup, iter_num,  dn_consensus, daltReads,  drefLens);
    hipDeviceSynchronize();
    CHECKERR1();

    hipMemcpy(BestScore, dBestScore, sizeof(int) * score_offset, hipMemcpyDeviceToHost);
    CHECKERR1();

    hipMemcpy(ScoreIndex, dScoreIndex, sizeof(int) * score_offset, hipMemcpyDeviceToHost);
    CHECKERR1();
}

int clean_cuda_prep(vector<BamRegionData *> &reads, int iter_num, int **ScoreIndex, int **BestScore, size_t &globalWorkSize, size_t &localWorkSize)
{

    int t_altReads = 0;
    int t_refLen = 0;
    int t_score = 0;
    int n_refLen = 0;
    for(int i = 0; i < iter_num; i++) {
        vector<Consensus *> finalConsensus;
        vector<Consensus *>::iterator itr;
        BamRegionData *rd = reads[i];
        ConsensusHashTable *altConsensus = rd->altConsensus;
        altConsensus->toArray(finalConsensus);
        vector<AlignedRead *>& altReads = rd->altReads;
        t_altReads += altReads.size();
        for(itr = finalConsensus.begin(); itr != finalConsensus.end(); itr++) {
            Consensus *consensus = *itr;
            t_refLen += consensus->strLen;
            t_score += altReads.size();
            n_refLen += 1;
        }
    }
    score_offset = t_score;
    if(t_altReads == 0 || t_refLen == 0 || t_score == 0) {
        return 0;
    }


    *ScoreIndex = (int *)malloc(sizeof(int) * t_score);
    *BestScore = (int *)malloc(sizeof(int) * t_score);
    byte *h_readSeq = (byte *)malloc(sizeof(byte) * MAX_ReadLength * t_altReads);
    byte *h_quals = (byte *)malloc(sizeof(byte) * MAX_ReadLength * t_altReads);
    byte *h_ref = (byte *)malloc(sizeof(byte) * t_refLen);
    int *ReadLength = (int *)malloc(sizeof(int) * t_altReads);
    int *originalAlignment = (int *)malloc(sizeof(int) * t_altReads);
    int *h_n_consensus = (int *)malloc(sizeof(int) * iter_num);
    int *h_altReads = (int *)malloc(sizeof(int) * iter_num);
    int *h_refLens = (int *)malloc(sizeof(int) * n_refLen);

    localWorkSize = 256;
    globalWorkSize = 1;
    int reads_off = 0, ref_off = 0, Consensus_off = 0;

    for(int i = 0; i < iter_num; i++) {
        vector<Consensus *> finalConsensus;
        vector<Consensus *>::iterator itr;
        BamRegionData *rd = reads[i];
        ReadBin *readsToClean = rd->rb;
        vector<AlignedRead *>& altReads = rd->altReads;
        ConsensusHashTable *altConsensus = rd->altConsensus;
        altConsensus->toArray(finalConsensus);
        int leftmostIndex = readsToClean->getStart();
        h_altReads[i] = altReads.size();

        if(altReads.size() > globalWorkSize) {
            globalWorkSize = altReads.size();
        }

        for(unsigned int j = 0; j < altReads.size(); j++) {
            AlignedRead *toTest = altReads[j];
            byte *readSeq = (*toTest).getReadBases();
            byte *quals = (*toTest).getBaseQualities();
            ReadLength[reads_off] = (*toTest).getReadLength();
            originalAlignment[reads_off] = (*toTest).getOriginalAlignmentStart() - leftmostIndex;
            memcpy(h_readSeq + reads_off * MAX_ReadLength, readSeq, sizeof(byte) * ReadLength[reads_off]);
            memcpy(h_quals + reads_off * MAX_ReadLength, quals, sizeof(byte) * ReadLength[reads_off]);
            if(originalAlignment[reads_off] > 512) {
                localWorkSize = 512;
            }
            reads_off += 1;
        }

        h_n_consensus[i] = finalConsensus.size();
        for(itr = finalConsensus.begin(); itr != finalConsensus.end(); itr++) {
            Consensus *consensus = *itr;
            memcpy(h_ref + ref_off, consensus->str, sizeof(byte) * consensus->strLen);
            ref_off += consensus->strLen;
            h_refLens[Consensus_off] = consensus->strLen;
            Consensus_off += 1;
        }
    }


    CHECKERR(hipMalloc((void **)&dreadSeq, sizeof(byte) * MAX_ReadLength * t_altReads));
    CHECKERR(hipMalloc((void **)&dquals, sizeof(byte) * MAX_ReadLength * t_altReads));
    CHECKERR(hipMalloc((void **)&dReadLength, sizeof(int) * t_altReads));
    CHECKERR(hipMalloc((void **)&doriginalAlignment, sizeof(int) * t_altReads));
    CHECKERR(hipMalloc((void **)&dref, sizeof(byte) * t_refLen));
    CHECKERR(hipMalloc((void **)&dn_consensus, sizeof(int) * iter_num));
    CHECKERR(hipMalloc((void **)&daltReads, sizeof(int) * iter_num));
    CHECKERR(hipMalloc((void **)&drefLens, sizeof(int) * n_refLen));

    CHECKERR(hipMalloc((void **)&dScoreIndex, sizeof(int) * t_score));
    CHECKERR(hipMalloc((void **)&dBestScore, sizeof(int) * t_score));

    hipMemcpy(dreadSeq, h_readSeq, sizeof(byte) * MAX_ReadLength * t_altReads, hipMemcpyHostToDevice);
    CHECKERR1();
    hipMemcpy(dquals, h_quals, sizeof(byte) * MAX_ReadLength * t_altReads, hipMemcpyHostToDevice);
    CHECKERR1();
    hipMemcpy(dReadLength, ReadLength, sizeof(int) * t_altReads, hipMemcpyHostToDevice);
    CHECKERR1();
    hipMemcpy(doriginalAlignment, originalAlignment, sizeof(int) * t_altReads, hipMemcpyHostToDevice);
    CHECKERR1();
    hipMemcpy(dref, h_ref, sizeof(byte) * t_refLen, hipMemcpyHostToDevice);
    CHECKERR1();
    hipMemcpy(dn_consensus, h_n_consensus, sizeof(int) * iter_num, hipMemcpyHostToDevice);
    CHECKERR1();
    hipMemcpy(daltReads, h_altReads, sizeof(int) * iter_num, hipMemcpyHostToDevice);
    CHECKERR1();
    hipMemcpy(drefLens, h_refLens, sizeof(int) * n_refLen, hipMemcpyHostToDevice);
    CHECKERR1();



    free(h_n_consensus);
    free(h_altReads);
    free(h_refLens);
    free(h_ref);
    free(h_readSeq);
    free(h_quals);
    free(ReadLength);
    free(originalAlignment);
    return 1;
}


